#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"
#include <iostream>
#include <stdio.h>

__device__ int getMax(int value1, int value2)
{
	return value1 > value2 ? value1 : value2;
}

__device__ int getMin(int value1, int value2)
{
	return value1 < value2 ? value1 : value2;
}

__device__ int clamp(int value, int minValue, int maxValue)
{
	return getMax( getMin( value, maxValue ), minValue );
}

__device__ int getPosition(int x, int y, int width, int margin, int pixelPosition)
{
	return (x + (y * width)) * margin + pixelPosition; 
}

__device__ void getMaxIndex(int array[], int size, int &maxValue, int &maxIndex)
{
	maxValue = -1; maxIndex = -1;
	for( int i = 0; i <= size; i++ )
	{
		if( maxValue < array[i] )
		{
			maxValue = array[i];
			maxIndex = i;
		}
	}
}

__global__ void setEdgeDetection( unsigned char* output_img, const unsigned char* input_img, int width, int height, int nbBlocks )
{
	int margin = 3;

	int lengthY = (int)(height/nbBlocks)+1;
	int startY = blockIdx.x * lengthY;
	int endY = blockIdx.x * lengthY + lengthY;

	if( endY > height )
		endY = height;
	
	int lengthX = (int)(width/blockDim.x)+1;
	int startX = threadIdx.x * lengthX;
	int endX = threadIdx.x * lengthX + lengthX;

	if( endX > width )
		endX = width;
	
	float kernel[9] = {
		1.0, 0.0, -1.0,
		0.0, 0.0, 0.0,
		-1.0, 0.0, 1.0 
	};
	float kernelDiv = 1.0f;

	for( int x = startX; x < endX; x++ )
	{
		for( int y = startY; y < endY; y++ )
		{
			int currentIndex = getPosition(x, y, width, margin, 0);
			float countR = 0;
			float countG = 0;
			float countB = 0;

			int n = 0;

			for( int i = -1; i <= 1; i++ )
			{
				if( y+i < 0 || y+i >= height )
					continue;

				for( int j = -1; j <= 1; j++)
				{
					if( x+i < 0 || x+i >= width )
						continue;

					int currentIndex2 = getPosition(x+j, y+i, width, margin, 0);
					countR += input_img[currentIndex2] / 255.0f * kernel[n];
					countG += input_img[currentIndex2+1] / 255.0f * kernel[n];
					countB += input_img[currentIndex2+2] / 255.0f * kernel[n];
					n++;
				}
			}

			countR *= kernelDiv;
			countG *= kernelDiv;
			countB *= kernelDiv;

			output_img[currentIndex] = clamp(255 - countR * 255 * 20, 0, 255);
			output_img[currentIndex+1] = clamp(255 - countG * 255 * 20, 0,255);  
			output_img[currentIndex+2] = clamp(255 - countB * 255 * 20, 0, 255);
		}
	}	
}

__global__ void setOilFilter(unsigned char* output_img, const unsigned char* input_img, int width, int height, int radius, int intensity, int nbBlocks)
{
	int margin = 3;

	int lengthY = (int)(height/nbBlocks)+1;
	int startY = blockIdx.x * lengthY;
	int endY = blockIdx.x * lengthY + lengthY;

	if( endY > height )
		endY = height;
	
	int lengthX = (int)(width/blockDim.x)+1;
	int startX = threadIdx.x * lengthX;
	int endX = threadIdx.x * lengthX + lengthX;

	if( endX > width )
		endX = width;

	for( int x = startX; x < endX; x++ )
	{
		for( int y = startY; y < endY; y++)
		{
			int currentIndex = getPosition(x, y, width, margin, 0);
			int intensityCount[255] = {0};
			int intensityR[255] = {0};
			int intensityG[255] = {0};
			int intensityB[255] = {0};

			for( int i = -radius; i <= radius; i++ )
			{
				if( y+i < 0 || y+i >= height )
					continue;

				for( int j = -radius; j <= radius; j++ )
				{
					if( x+j < 0 || x+j >= width )
						continue;

					int currentIndex2 = getPosition(x+j, y+i, width, margin, 0);
					int R = input_img[currentIndex2];
					int G = input_img[currentIndex2+1];
					int B = input_img[currentIndex2+2];

					int currentIntensity = (((R+G+B)/3.0)*intensity)/255.0;
				
					intensityCount[currentIntensity]++;
					intensityR[currentIntensity] += R;
					intensityG[currentIntensity] += G;
					intensityB[currentIntensity] += B;
				}	
			}
			int maxValue = 0; int maxIndex = 0;
			
			getMaxIndex(intensityCount, intensity, maxValue, maxIndex);
			output_img[currentIndex] = clamp(intensityR[maxIndex]/maxValue, 0, 255);
			output_img[currentIndex+1] = clamp(intensityG[maxIndex]/maxValue, 0, 255);
			output_img[currentIndex+2] = clamp(intensityB[maxIndex]/maxValue, 0, 255);
		}
	}
}

__global__ void addEffect( unsigned char* output_img, unsigned char* input_img, int width, int height, int nbBlocks)
{
	int lengthY = (int)(height/nbBlocks)+1;
	int startY = blockIdx.x * lengthY;
	int endY = blockIdx.x * lengthY + lengthY;

	if( endY > height )
		endY = height;
	
	int lengthX = (int)(width/blockDim.x)+1;
	int startX = threadIdx.x * lengthX;
	int endX = threadIdx.x * lengthX + lengthX;

	if( endX > width )
		endX = width;

	for( int x = startX; x < endX; x++ )
	{
		for( int y = startY; y < endY; y++ )
		{
			int currentIndex = getPosition(x, y, width, 3, 0);
			if( (input_img[currentIndex] + input_img[currentIndex+1] + input_img[currentIndex+2])/3 < 20)
			{
				output_img[currentIndex] = input_img[currentIndex];
				output_img[currentIndex+1] = input_img[currentIndex+1];
				output_img[currentIndex+2] = input_img[currentIndex+2];

				for( int i = -4; i <= 4; i++ )
				{
					for( int j = -4; j <= 4; j++ )
					{
						if( x+i < 0 || x+i > width || y+j < 0 || y+j > height )
							continue;

						int neighbourIndex = getPosition( x+i, y+j, width, 3, 0);

						if( neighbourIndex < 0 || neighbourIndex + 2 > width*height*3)
							continue;
					
						output_img[neighbourIndex] = 0;
						output_img[neighbourIndex+1] = 0;
						output_img[neighbourIndex+2] = 0;
					}
					
				}
			}
		}
		
	}

}

int main()
{
	int width, height, n;
	unsigned char* sourceImg = stbi_load("Photos/01.jpg", &width, &height, &n, 3);
	int nbBlocks = 13; int nbThreads = 1024;
	
	unsigned char* inputImg, *inputImg2, *outputImg, *tmpOutput;
	hipMalloc((void**) &inputImg, width * height * n * sizeof(unsigned char));
	hipMemcpy(inputImg, sourceImg, width * height * n * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMallocManaged(&outputImg, width * height * n * sizeof(unsigned char));
	hipMallocManaged(&tmpOutput, width * height * n * sizeof(unsigned char));
	
	// OIL FILTER
	setOilFilter<<<nbBlocks,nbThreads>>>(outputImg, inputImg, width, height, 10, 20, nbBlocks);
	hipDeviceSynchronize();

	hipMalloc((void**) &inputImg2, width * height * n * sizeof(unsigned char));
	hipMemcpy(inputImg2, outputImg, width * height * n * sizeof(unsigned char), hipMemcpyDeviceToDevice);

	// EDGE DETECTION
	setEdgeDetection<<<nbBlocks,nbThreads>>>(tmpOutput, inputImg2, width, height, nbBlocks);
	hipDeviceSynchronize();

	// FUSION
	addEffect<<<nbBlocks,nbThreads>>>(outputImg, tmpOutput, width, height, nbBlocks);
	hipDeviceSynchronize();

	stbi_write_png("exempleCuda.png", width, height, n, outputImg, n*width);
	
	hipFree(inputImg2);
	hipFree(tmpOutput);
	hipFree(outputImg);
	hipFree(inputImg);

	return 0;
}
